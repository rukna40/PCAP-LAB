#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void add(int *a, int *b, int *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main(void)
{
    int n;

    printf("Enter n: ");
    scanf("%d", &n);

    int a[n], b[n], c[n];

    printf("Enter elements for 1st array:\n");
    for (int i = 0; i < n; i++)
        scanf("%d", &a[i]);

    printf("Enter elements for 2nd array:\n");
    for (int i = 0; i < n; i++)
        scanf("%d", &b[i]);

    int *d_a, *d_b, *d_c;

    int s = sizeof(int) * n;

    hipMalloc((void **)&d_a, s);
    hipMalloc((void **)&d_b, s);
    hipMalloc((void **)&d_c, s);

    hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, s, hipMemcpyHostToDevice);

    add<<<1, n>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);

    printf("Vector (1,n): \n");
    for (int i = 0; i < n; i++)
        printf("%d ", c[i]);
    printf("\n");

    add<<<n, 1>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);

    printf("Vector (n,1): \n");
    for (int i = 0; i < n; i++)
        printf("%d ", c[i]);
    printf("\n");

    dim3 dimGrid((int)ceil(n / 256.0), 1, 1); 
    dim3 dimBlock(256, 1, 1); 

    add<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);

    printf("Vector (%d, 256): \n", dimGrid.x);
    for (int i = 0; i < n; i++)
        printf("%d ", c[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}