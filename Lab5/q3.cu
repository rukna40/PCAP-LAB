#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"


__global__ void sinArray(float *a, float *b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        b[i] = sinf(a[i]);
}

int main(void)
{
    int n;

    printf("Enter n: ");
    scanf("%d", &n);

    float *a = (float *)malloc(n * sizeof(float));
    float *b = (float *)malloc(n * sizeof(float));

    printf("Enter elements for array:\n");
    for (int i = 0; i < n; i++)
        scanf("%f", &a[i]);

    float *d_a, *d_b;

    int s = sizeof(float) * n;

    hipMalloc((void **)&d_a, s);
    hipMalloc((void **)&d_b, s);

    hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);

    sinArray<<<1, n>>>(d_a, d_b, n);
    hipMemcpy(b, d_b, s, hipMemcpyDeviceToHost);

    printf("Sin Array: \n");
    for (int i = 0; i < n; i++)
        printf("%f ", b[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
