#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void repeat_string(char* input, char* output, int str_len, int total_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < total_len) {
        int repeat_idx = idx % str_len;
        output[idx] = input[repeat_idx];
    }
}

int main() {
    const char* input_string = "PCAP";
    int str_len = strlen(input_string);
    int total_len = 9;

    char* d_input;
    char* d_output;

    hipMalloc((void**)&d_input, str_len + 1);
    hipMalloc((void**)&d_output, total_len + 1);

    hipMemcpy(d_input, input_string, str_len + 1, hipMemcpyHostToDevice);

    dim3 dimGrid(width, 1, 1);
    dim3 dimBlock(256, 1, 1);

    repeat_string<<<dimGrid, dimBlock>>>(d_input, d_output, str_len, total_len);

    char output_string[total_len + 1];
    hipMemcpy(output_string, d_output, total_len + 1, hipMemcpyDeviceToHost);

    printf("Input string: %s\n", input_string);
    printf("Output string: %s\n", output_string);

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
