#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void count_word(char* sentence, char* word, int* count, int sentence_len, int word_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < sentence_len - word_len + 1) {
        bool match = true;
        for (int i = 0; i < word_len; ++i) {
            if (sentence[idx + i] != word[i]) {
                match = false;
                break;
            }
        }

        if (match) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    const char* sentence = "This is a test sentence, and this is a test";
    const char* word = "test";
    
    int sentence_len = strlen(sentence);
    int word_len = strlen(word);
    int count = 0;

    char* d_sentence;
    char* d_word;
    int* d_count;

    hipMalloc((void**)&d_sentence, sentence_len + 1);
    hipMalloc((void**)&d_word, word_len + 1);
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_sentence, sentence, sentence_len + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, word_len + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(width, 1, 1);
    dim3 dimBlock(256, 1, 1);

    count_word<<<dimGrid, dimBlock>>>(d_sentence, d_word, d_count, sentence_len, word_len);

    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", word, count);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}
