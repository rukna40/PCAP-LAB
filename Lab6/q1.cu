#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void conv(int *N, int width, int *M, int mask_width, int *P)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (i >= (mask_width / 2) - 1 && i <= width - mask_width / 2)
    {
        for (int j = -mask_width / 2; j <= mask_width / 2; j++)
            sum += N[i + j] * M[j + mask_width / 2];
        P[i] = sum;
    }
}

int main()
{
    int width, mask_width;

    printf("Enter width: ");
    scanf("%d", &width);

    printf("Enter mask width: ");
    scanf("%d", &mask_width);

    int N[width], M[mask_width], P[width];

    printf("Enter elements for array N:\n");
    for (int i = 0; i < width; i++)
        scanf("%d", &N[i]);

    printf("Enter elements for mask array M:\n");
    for (int i = 0; i < mask_width; i++)
        scanf("%d", &M[i]);

    int *d_n, *d_m, *d_p;
    int n = sizeof(int) * width, m = sizeof(int) * mask_width;

    hipMalloc((void **)&d_n, n);
    hipMalloc((void **)&d_m, m);
    hipMalloc((void **)&d_p, n);

    hipMemcpy(d_n, N, n, hipMemcpyHostToDevice);
    hipMemcpy(d_m, M, m, hipMemcpyHostToDevice);

    dim3 dimGrid(width, 1, 1);
    dim3 dimBlock(256, 1, 1);

    conv<<<dimGrid, dimBlock>>>(d_n, width, d_m, mask_width, d_p);

    hipMemcpy(P, d_p, n, hipMemcpyDeviceToHost);

    printf("Output:\n");
    for (int i = 0; i < width; i++)
        printf("%d ", P[i]);
    printf("\n");

    hipFree(d_n);
    hipFree(d_m);
    hipFree(d_p);

    return 0;
}