#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void odd(int *arr, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % 2 == 1 && i < n - 1)
    {
        if (arr[i] > arr[i + 1])
        {
            int temp = arr[i];
            arr[i] = arr[i + 1];
            arr[i + 1] = temp;
        }
    }
}

__global__ void even(int *arr, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i % 2 == 0 && i < n - 1)
    {
        if (arr[i] > arr[i + 1])
        {
            int temp = arr[i];
            arr[i] = arr[i + 1];
            arr[i + 1] = temp;
        }
    }
}

int main()
{
    int n;

    printf("Enter n: ");
    scanf("%d", &n);

    int *arr = (int *)malloc(sizeof(int) * n);

    printf("Enter elements for array:\n");
    for (int i = 0; i < n; i++)
    {
        scanf("%d", &arr[i]);
    }

    int *d_arr;

    hipMalloc((void **)&d_arr, sizeof(int) * n);

    hipMemcpy(d_arr, arr, sizeof(int) * n, hipMemcpyHostToDevice);

    dim3 dimGrid(n, 1, 1);
    dim3 dimBlock(1, 1, 1);

    for (int i = 0; i < n; i++)
    {
        even<<<dimGrid, dimBlock>>>(d_arr, n);
        odd<<<dimGrid, dimBlock>>>(d_arr, n);
    }

    hipMemcpy(arr, d_arr, sizeof(int) * n, hipMemcpyDeviceToHost);

    printf("Sorted Output:\n");
    for (int i = 0; i < n; i++)
    {
        printf("%d ", arr[i]);
    }
    printf("\n");

    hipFree(d_arr);

    free(arr);

    return 0;
}
