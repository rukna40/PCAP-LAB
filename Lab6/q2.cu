#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void selection_sort(int *arr, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n) {
        int pos = 0;
        for (int j = 0; j < n; j++) {
            if (arr[j] < arr[i]) {
                pos++;
            }
        }

        arr[pos] = arr[i];
    }
}

int main()
{
    int n;

    printf("Enter n: ");
    scanf("%d", &n);

    int arr[n];

    printf("Enter elements for array:\n");
    for (int i = 0; i < n; i++)
        scanf("%d", &arr[i]);

    int *d_arr;

    hipMalloc((void **)&d_arr, sizeof(int)*n);

    hipMemcpy(d_arr, arr, sizeof(int)*n, hipMemcpyHostToDevice);

    dim3 dimGrid(n, 1, 1);
    dim3 dimBlock(1, 1, 1);

    selection_sort<<<dimGrid, dimBlock>>>(d_arr, n);

    hipMemcpy(arr, d_arr, sizeof(int) * n, hipMemcpyDeviceToHost);

    printf("Output:\n");
    for (int i = 0; i < n; i++)
        printf("%d ", arr[i]);
    printf("\n");

    hipFree(d_arr);
    return 0;
}