#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void csr_mv(int *r, int *c, float *v, float *vec, float *res, int rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = 0;
        for (int i = r[row]; i < r[row + 1]; i++) {
            sum += v[i] * vec[c[i]];
        }
        res[row] = sum;
    }
}

int main() {
    int rows, cols, nze, *row_ptr, *col_idx;
    float *values, *vec, *res;

    printf("Enter number of rows, columns, non-zero elements: ");
    scanf("%d %d %d", &rows, &cols, &nze);

    values = (float *)malloc(nze * sizeof(float)); 
    row_ptr = (int *)calloc(rows + 1, sizeof(int));
    col_idx = (int *)malloc(nze * sizeof(int));
    
    printf("Enter sparse matrix (row col value):\n");
    for (int i = 0; i < nze; i++) {
        int row, col;
        float value;
        scanf("%d %d %f", &row, &col, &value);
        row_ptr[row + 1]++; 
        values[i] = value; 
        col_idx[i] = col; 
    }

    for (int i = 1; i <= rows; i++) {
        row_ptr[i] += row_ptr[i - 1];
    }

    vec = (float *)malloc(cols * sizeof(float));
    res = (float *)calloc(rows, sizeof(float));

    printf("Enter vector elements:\n");
    for (int i = 0; i < cols; i++) {
        scanf("%f", &vec[i]);
    }

    int *dr, *dc;
    float *dv, *dvec, *dres;

    hipMalloc(&dr, (rows + 1) * sizeof(int));
    hipMalloc(&dc, nze * sizeof(int));
    hipMalloc(&dv, nze * sizeof(float));
    hipMalloc(&dvec, cols * sizeof(float));
    hipMalloc(&dres, rows * sizeof(float));

    hipMemcpy(dr, row_ptr, (rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dc, col_idx, nze * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dv, values, nze * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dvec, vec, cols * sizeof(float), hipMemcpyHostToDevice);

    int bs = 256;
    int gs = (rows + bs - 1) / bs;
    csr_mv<<<gs, bs>>>(dr, dc, dv, dvec, dres, rows);

    hipMemcpy(res, dres, rows * sizeof(float), hipMemcpyDeviceToHost);

    printf("\nResult:\n");
    for (int i = 0; i < rows; i++) {
        printf("%f ", res[i]);
    }
    printf("\n");

    hipFree(dr);
    hipFree(dc);
    hipFree(dv);
    hipFree(dvec);
    hipFree(dres);

    free(row_ptr);
    free(col_idx);
    free(values);
    free(vec);
    free(res);

    return 0;
}