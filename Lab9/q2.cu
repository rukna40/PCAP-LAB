#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void powers(int *d_A, int M, int N)
{
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    if (row < M)
    {
        for (int col = 0; col < N; col++)
        {
            if (row == 0)
                d_A[row * N + col] = d_A[row * N + col];
            else
                d_A[row * N + col] = pow(d_A[row * N + col], row + 1);
        }
    }
}

int main()
{
    int M, N;

    printf("Enter the number of rows (M): ");
    scanf("%d", &M);
    printf("Enter the number of columns (N): ");
    scanf("%d", &N);

    int *h_A = (int *)malloc(M * N * sizeof(int));

    printf("Enter the elements of the matrix (%d elements):\n", M * N);
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            scanf("%d", &h_A[i * N + j]);
        }
    }

    int *d_A;
    size_t size = M * N * sizeof(int);
    hipMalloc((void **)&d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    powers<<<M, 1>>>(d_A, M, N);

    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    hipFree(d_A);

    printf("Modified Matrix:\n");
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", h_A[i * N + j]);
        }
        printf("\n");
    }

    free(h_A);

    return 0;
}