#include <stdio.h>
#include <hip/hip_runtime.h>


__device__ int ones_complement(int n) {
    int bin=0;
    int inc=1;
    for(int i = n; i > 0; i /= 2) {
        bin += (i%2 == 0)*inc;
        inc *= 10;
    }
    return bin;
}

__global__ void non_border_complement(int *mat, int *out, int rows, int cols) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;

    if(r < rows && c < cols) {
        int elem;
        if (r == 0 || r == rows-1 || c == 0 || c == cols-1) {
            elem = mat[r*cols + c];
        } else {
            elem = ones_complement(mat[r*cols + c]);
        }
        out[r*cols + c] = elem;
    }
}

int main() {
    int r, c;
    printf("Rows: ");
    scanf("%d", &r);
    printf("Columns: ");
    scanf("%d", &c);

    int mat[r][c], out[r][c];

    printf("Enter matrix:\n");
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            scanf("%d", &mat[i][j]);
        }
    }

    int *d_mat, *d_out;
    hipMalloc((void **)&d_mat, r * c * sizeof(int));
    hipMalloc((void **)&d_out, r * c * sizeof(int));

    hipMemcpy(d_mat, mat, r * c * sizeof(int), hipMemcpyHostToDevice);

    non_border_complement<<<dim3(ceil(r/32.0), ceil(c/32.0)), dim3(32, 32)>>>(d_mat, d_out, r, c);
    hipMemcpy(out, d_out, r * c * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            printf("%d ", out[i][j]);
        }
        printf("\n");
    }

    hipFree(d_mat);
    hipFree(d_out);
    return 0;
}