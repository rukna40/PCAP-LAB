#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void multiplyrow(int n, int m, int p, int *mat1, int *mat2, int *res)
{
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    if (row < n)
    {
        for (int col = 0; col < p; col++)
        {
            int sum = 0;
            for (int k = 0; k < m; k++)
            {
                sum += mat1[row * m + k] * mat2[k * p + col];
            }
            res[row * p + col] = sum;
        }
    }
}

__global__ void multiplycol(int n, int m, int p, int *mat1, int *mat2, int *res)
{
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    if (col < n)
    {
        for (int row = 0; row < n; row++)
        {
            int sum = 0;
            for (int k = 0; k < m; k++)
            {
                sum += mat1[row * m + k] * mat2[k * p + col];
            }
            res[row * p + col] = sum;
        }
    }
}

__global__ void multiplyele(int n, int m, int p, int *mat1, int *mat2, int *res)
{
    int idx = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx < n * p)
    {
        int row = idx / p;
        int col = idx % p;

        int sum = 0;

        for (int k = 0; k < m; k++)
        {
            sum += mat1[row * m + k] * mat2[k * p + col];
        }
        res[row * p + col] = sum;
    }
}

int main()
{
    int n, m, p;
    printf("Enter n: ");
    scanf("%d", &n);
    printf("Enter m: ");
    scanf("%d", &m);
    printf("Enter p: ");
    scanf("%d", &p);

    int mat1[n][m], mat2[m][p], res[n][p];

    printf("Enter elements for matrix 1:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            scanf("%d", &mat1[i][j]);
        }
    }

    printf("Enter elements for matrix 2:\n");
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < p; j++)
        {
            scanf("%d", &mat2[i][j]);
        }
    }

    int *d_m1, *d_m2, *d_res;
    hipMalloc((void **)&d_m1, sizeof(int) * n * m);
    hipMalloc((void **)&d_m2, sizeof(int) * m * p);
    hipMalloc((void **)&d_res, sizeof(int) * n * p);

    hipMemcpy(d_m1, mat1, sizeof(int) * n * m, hipMemcpyHostToDevice);
    hipMemcpy(d_m2, mat2, sizeof(int) * m * p, hipMemcpyHostToDevice);

    multiplyrow<<<n, m>>>(n, m, p, d_m1, d_m2, d_res);

    hipMemcpy(res, d_res, sizeof(int) * n * p, hipMemcpyDeviceToHost);

    printf("Row wise result:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < p; j++)
        {
            printf("%d\t", res[i][j]);
        }
        printf("\n");
    }

    multiplycol<<<n, m>>>(n, m, p, d_m1, d_m2, d_res);

    hipMemcpy(res, d_res, sizeof(int) * n * p, hipMemcpyDeviceToHost);

    printf("Column wise Result:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < p; j++)
        {
            printf("%d\t", res[i][j]);
        }
        printf("\n");
    }

    multiplyele<<<n, m>>>(n, m, p, d_m1, d_m2, d_res);

    hipMemcpy(res, d_res, sizeof(int) * n * p, hipMemcpyDeviceToHost);

    printf("Element wise result:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < p; j++)
        {
            printf("%d\t", res[i][j]);
        }
        printf("\n");
    }

    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_res);

    return 0;
}