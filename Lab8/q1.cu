#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void addrow(int n, int m, int *mat1, int *mat2, int *res)
{
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    if (row < n)
    {
        for (int j = 0; j < m; j++)
        {
            res[row * m + j] = mat1[row * m + j] + mat2[row * m + j];
        }
    }
}

__global__ void addcol(int n, int m, int *mat1, int *mat2, int *res)
{
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    if (col < m)
    {
        for (int i = 0; i < n; i++)
        {
            res[i * m + col] = mat1[i * m + col] + mat2[i * m + col];
        }
    }
}

__global__ void addele(int n, int m, int *mat1, int *mat2, int *res)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n * m)
    {
        res[idx] = mat1[idx] + mat2[idx];
    }
}

int main()
{
    int n, m;
    printf("Enter n: ");
    scanf("%d", &n);
    printf("Enter m: ");
    scanf("%d", &m);

    int mat1[n][m], mat2[n][m], res[n][m];

    printf("Enter elements for matrix 1:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            scanf("%d", &mat1[i][j]);
        }
    }

    printf("Enter elements for matrix 2:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            scanf("%d", &mat2[i][j]);
        }
    }

    int *d_m1, *d_m2, *d_res;
    hipMalloc((void **)&d_m1, sizeof(int) * n * m);
    hipMalloc((void **)&d_m2, sizeof(int) * n * m);
    hipMalloc((void **)&d_res, sizeof(int) * n * m);

    hipMemcpy(d_m1, mat1, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_m2, mat2, sizeof(int) * m * n, hipMemcpyHostToDevice);

    addrow<<<n, m>>>(n, m, d_m1, d_m2, d_res);

    hipMemcpy(res, d_res, sizeof(int) * m * n, hipMemcpyDeviceToHost);

    printf("Row wise result:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            printf("%d\t", res[i][j]);
        }
        printf("\n");
    }

    addcol<<<n, m>>>(n, m, d_m1, d_m2, d_res);

    hipMemcpy(res, d_res, sizeof(int) * m * n, hipMemcpyDeviceToHost);

    printf("Column wise Result:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            printf("%d\t", res[i][j]);
        }
        printf("\n");
    }

    addele<<<n, m>>>(n, m, d_m1, d_m2, d_res);

    hipMemcpy(res, d_res, sizeof(int) * m * n, hipMemcpyDeviceToHost);

    printf("Element wise result:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            printf("%d\t", res[i][j]);
        }
        printf("\n");
    }

    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_res);

    return 0;
}